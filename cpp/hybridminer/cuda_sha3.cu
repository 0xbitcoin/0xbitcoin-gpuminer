#include "hip/hip_runtime.h"
#include <unistd.h>
#include <time.h>

#include <sys/time.h>

#define LOOP_IN_GPU_OPTIMIZATION 10000
#include <hiprand.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

/*

Author: Mikers
date march 4, 2018 for 0xbitcoin dev

based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu

 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */


 #include "cudasolver.h"

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

void gpu_init();
void runBenchmarks();
char *read_in_messages();
int gcd(int a, int b);

// updated message the gpu_init() function
int clock_speed;
int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;
int h_done[1] = {0};

int num_messages;
const int digest_size = 256;
const int digest_size_bytes = digest_size / 8;
const size_t str_length = 7;	//change for different sizes

hipEvent_t start, stop;

#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))

__device__ const char *chars =
    " !\"#$%&\'()*+'-./0123456789:;<=>?@ABCDEFGHIJKLMOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~";

__device__ const uint64_t RC[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ const int r[24] = {
    1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14,
    27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
};

__device__ const int piln[24] = {
    10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4,
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1
};




__device__ int compare_hash(unsigned char *target, unsigned char *hash, int length)
{

	int i =0;
	for (i = 0; i < length; i++)
	{
		if(hash[i] != target[i])break;
	}
	return (unsigned char)(hash[i]) < (unsigned char)(target[i]);


}

__device__ void keccak256(uint64_t state[25])
{
    uint64_t temp, C[5];
	int j;

    for (int i = 0; i < 24; i++) {
        // Theta
		// for i = 0 to 5
		//    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
		C[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
		C[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
		C[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
		C[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
		C[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];

		// for i = 0 to 5
		//     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
		//     for j = 0 to 25, j += 5
		//          state[j + i] ^= temp;
		temp = C[4] ^ ROTL64(C[1], 1);
		state[0] ^= temp;
		state[5] ^= temp;
		state[10] ^= temp;
		state[15] ^= temp;
		state[20] ^= temp;

		temp = C[0] ^ ROTL64(C[2], 1);
		state[1] ^= temp;
		state[6] ^= temp;
		state[11] ^= temp;
		state[16] ^= temp;
		state[21] ^= temp;

		temp = C[1] ^ ROTL64(C[3], 1);
		state[2] ^= temp;
		state[7] ^= temp;
		state[12] ^= temp;
		state[17] ^= temp;
		state[22] ^= temp;

		temp = C[2] ^ ROTL64(C[4], 1);
		state[3] ^= temp;
		state[8] ^= temp;
		state[13] ^= temp;
		state[18] ^= temp;
		state[23] ^= temp;

		temp = C[3] ^ ROTL64(C[0], 1);
		state[4] ^= temp;
		state[9] ^= temp;
		state[14] ^= temp;
		state[19] ^= temp;
		state[24] ^= temp;

        // Rho Pi
		// for i = 0 to 24
		//     j = piln[i];
		//     C[0] = state[j];
		//     state[j] = ROTL64(temp, r[i]);
		//     temp = C[0];
		temp = state[1];
		j = piln[0];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[0]);
		temp = C[0];

		j = piln[1];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[1]);
		temp = C[0];

		j = piln[2];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[2]);
		temp = C[0];

		j = piln[3];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[3]);
		temp = C[0];

		j = piln[4];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[4]);
		temp = C[0];

		j = piln[5];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[5]);
		temp = C[0];

		j = piln[6];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[6]);
		temp = C[0];

		j = piln[7];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[7]);
		temp = C[0];

		j = piln[8];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[8]);
		temp = C[0];

		j = piln[9];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[9]);
		temp = C[0];

		j = piln[10];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[10]);
		temp = C[0];

		j = piln[11];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[11]);
		temp = C[0];

		j = piln[12];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[12]);
		temp = C[0];

		j = piln[13];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[13]);
		temp = C[0];

		j = piln[14];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[14]);
		temp = C[0];

		j = piln[15];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[15]);
		temp = C[0];

		j = piln[16];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[16]);
		temp = C[0];

		j = piln[17];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[17]);
		temp = C[0];

		j = piln[18];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[18]);
		temp = C[0];

		j = piln[19];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[19]);
		temp = C[0];

		j = piln[20];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[20]);
		temp = C[0];

		j = piln[21];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[21]);
		temp = C[0];

		j = piln[22];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[22]);
		temp = C[0];

		j = piln[23];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[23]);
		temp = C[0];

        //  Chi
		// for j = 0 to 25, j += 5
		//     for i = 0 to 5
		//         C[i] = state[j + i];
		//     for i = 0 to 5
		//         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
		C[0] = state[0];
		C[1] = state[1];
		C[2] = state[2];
		C[3] = state[3];
		C[4] = state[4];

		state[0] ^= (~C[1]) & C[2];
		state[1] ^= (~C[2]) & C[3];
		state[2] ^= (~C[3]) & C[4];
		state[3] ^= (~C[4]) & C[0];
		state[4] ^= (~C[0]) & C[1];

		C[0] = state[5];
		C[1] = state[6];
		C[2] = state[7];
		C[3] = state[8];
		C[4] = state[9];

		state[5] ^= (~C[1]) & C[2];
		state[6] ^= (~C[2]) & C[3];
		state[7] ^= (~C[3]) & C[4];
		state[8] ^= (~C[4]) & C[0];
		state[9] ^= (~C[0]) & C[1];

		C[0] = state[10];
		C[1] = state[11];
		C[2] = state[12];
		C[3] = state[13];
		C[4] = state[14];

		state[10] ^= (~C[1]) & C[2];
		state[11] ^= (~C[2]) & C[3];
		state[12] ^= (~C[3]) & C[4];
		state[13] ^= (~C[4]) & C[0];
		state[14] ^= (~C[0]) & C[1];

		C[0] = state[15];
		C[1] = state[16];
		C[2] = state[17];
		C[3] = state[18];
		C[4] = state[19];

		state[15] ^= (~C[1]) & C[2];
		state[16] ^= (~C[2]) & C[3];
		state[17] ^= (~C[3]) & C[4];
		state[18] ^= (~C[4]) & C[0];
		state[19] ^= (~C[0]) & C[1];

		C[0] = state[20];
		C[1] = state[21];
		C[2] = state[22];
		C[3] = state[23];
		C[4] = state[24];

		state[20] ^= (~C[1]) & C[2];
		state[21] ^= (~C[2]) & C[3];
		state[22] ^= (~C[3]) & C[4];
		state[23] ^= (~C[4]) & C[0];
		state[24] ^= (~C[0]) & C[1];

        //  Iota
        state[0] ^= RC[i];
    }
}

__device__ void keccak(const char *message, int message_len, unsigned char *output, int output_len)
{
    uint64_t state[25];
    uint8_t temp[144];
    int rsize = 136;
    int rsize_byte = 17;

    memset(state, 0, sizeof(state));

    for ( ; message_len >= rsize; message_len -= rsize, message += rsize) {
        for (int i = 0; i < rsize_byte; i++) {
            state[i] ^= ((uint64_t *) message)[i];
		}
        keccak256(state);
    }

    // last block and padding
    memcpy(temp, message, message_len);
    temp[message_len++] = 1;
    memset(temp + message_len, 0, rsize - message_len);
    temp[rsize - 1] |= 0x80;

    for (int i = 0; i < rsize_byte; i++) {
        state[i] ^= ((uint64_t *) temp)[i];
	}

    keccak256(state);
    memcpy(output, state, output_len);
}

// hash length is 256 bits
__global__ void gpu_mine( unsigned char *challenge_hash, char * device_solution, int *done,  const unsigned char * hash_prefix, int now, int cnt)
{
    __shared__ char * message_all;
    __shared__ char * hash_all;
    if (threadIdx.x == 0) {
        size_t size = blockDim.x * 84;
        message_all = (char*)malloc(size);
        size = blockDim.x * 32;
        hash_all = (char*)malloc(size);
    }
    __syncthreads();

int tid = threadIdx.x + (blockIdx.x * blockDim.x);
char * message = &message_all[84*(threadIdx.x)];
char * hash =&hash_all[32*(threadIdx.x)];

int str_len = 84;

  hiprandState_t state;
  /* we have to initialize the state */
  hiprand_init(now, tid, cnt, &state);
	int len = 0;
	for(len = 0 ; len < 52; len++){
		message[len] = hash_prefix[len];
	}
for(int i =0; i<LOOP_IN_GPU_OPTIMIZATION;i++){

	for(len = 0; len < 32; len++) {
		char r = (char)hiprand(&state) % 256;
		message[52+len] = r;
	}



	const int output_len = 32;
	unsigned char output[output_len];
	keccak(&message[0], str_len, &output[0], output_len);

	if (compare_hash(&challenge_hash[0], &output[0], output_len))
	{
		if(done[0] != 1){
			done[0] = 1;
			memcpy(device_solution, message, str_len);
		}
		return;
	}

}
    // Ensure all threads complete before freeing 
    __syncthreads();

    // Only one thread may free the memory!
    if (threadIdx.x == 0)
{

  free(message_all);
	free(hash_all);
}
}



void stop_solving()
{
  h_done[0] = 1 ;
}


/**
 * Initializes the global variables by calling the hipGetDeviceProperties().
 */
void gpu_init()
{
    hipDeviceProp_t device_prop;
    int device_count, block_size;

    hipGetDeviceCount(&device_count);
    if (device_count != 1) {
        printf("Only want to test a single GPU, exiting...\n");
        exit(EXIT_FAILURE);
    }

    if (hipGetDeviceProperties(&device_prop, 0) != hipSuccess) {
        printf("Problem getting properties for device, exiting...\n");
        exit(EXIT_FAILURE);
    }

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = 128;//max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = max_threads_per_mp / block_size;
    number_blocks = block_size * number_multi_processors ;
    clock_speed = (int) (device_prop.memoryClockRate * 1000 * 1000);    // convert from GHz to hertz
}

int gcd(int a, int b) {
    return (a == 0) ? b : gcd(b % a, a);
}


unsigned char * find_message(const char * challenge_target, const char * hash_prefix) // can accept challenge
{


    h_done[0] = 0;



		int *d_done;
		char *device_solution;

		unsigned char * d_challenge_hash;
		unsigned char * d_hash_prefix;

		hipMalloc((void**) &d_done, sizeof(int));
		hipMalloc((void**) &device_solution, 84); // solution
		hipMalloc((void**) &d_challenge_hash, 32);

		hipMalloc((void**) &d_hash_prefix, 52);

		hipMemcpy(d_done, h_done, sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(d_challenge_hash, challenge_target, 32, hipMemcpyHostToDevice);
		hipMemcpy(d_hash_prefix, hash_prefix, 52, hipMemcpyHostToDevice);

		cudaThreadSetLimit(hipLimitMallocHeapSize,2*(84*number_blocks*number_threads + 32*number_blocks*number_threads));
		int now = (int)time(0);
		unsigned long long cnt = 0;
  struct timeval t0;
  struct timeval t1;




gettimeofday(&t0, 0);


		while (!h_done[0]) {
			gpu_mine<<<number_blocks, number_threads>>>( d_challenge_hash, device_solution, d_done, d_hash_prefix, now,cnt);
			hipError_t cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess) {
				h_done[0] = 1;

        cout << cudaerr;
				printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
			}
        cnt+=number_threads*number_blocks*LOOP_IN_GPU_OPTIMIZATION;
if(time(0)!=now)

/* ... */
gettimeofday(&t1, 0);
long elapsed = (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;



fprintf(stderr,"Total Hashes: %u\tHash Rate:%f MH/s\n", cnt, (float(cnt)/float(elapsed)));

			hipMemcpy(h_done, d_done, sizeof(int), hipMemcpyDeviceToHost);
		}

	unsigned	 char * h_message = (unsigned char*)malloc(84);
		hipMemcpy(h_message, device_solution, 84, hipMemcpyDeviceToHost);
    FILE * fp;
    fp = fopen ("out.binary", "wb") ;
    fwrite(h_message , 84, 1 , fp );
		fclose(fp);
    fprintf(stderr,"Total hashes: %u\n", cnt);

	/*
   printf("MIKERS ANSWER IS : ");
		for (int j = 52; j < 84; j++)
		{
		      printf("%02x",(unsigned char) h_message[j]);
		}
		printf("\n");
*/

		hipFree(d_done);
		hipFree(device_solution);
		hipFree(d_challenge_hash);

		hipFree(d_hash_prefix);
    return h_message;
}

/**
 * Main method, initializes the global variables, calls the kernels, and prints the results.
 */
int init(int argc, char **argv)
{


	char * hash_prefix_filename = argv[1];
	char * challenge = argv[2]; // challenge is the target
	char  hash_prefix[53];


        FILE *f = fopen(hash_prefix_filename, "r");
	fread(&hash_prefix, 52, 1, f);

	hash_prefix[52]='\0';
	srand(time(0));

	char  challenge_target[32];

        FILE *fc = fopen(challenge, "r");
	fread(&challenge_target, 32, 1, fc);

	gpu_init();

	find_message(challenge_target, hash_prefix);

	return EXIT_SUCCESS;
}
